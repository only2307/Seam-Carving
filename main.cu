
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define FILTER_WIDTH 3
__constant__ float dc_filter1[FILTER_WIDTH * FILTER_WIDTH];
__constant__ float dc_filter2[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

char *concatStr(const char *s1, const char *s2)
{
    char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

uint8_t min_e_idx(int e1, int e2, int e3)
{
    if (e2 <= e1 && e2 <= e3)
    {
        return 1;
    }
    else if (e3 <= e1 && e3 <= e2)
    {
        return 2;
    }
    else if (e1 <= e2 && e1 <= e3)
    {
        return 0;
    }
    return 255;
}

void readPnm(char *fileName, int &numChannels, int &width, int &height, uint8_t *&pixels)
{
    FILE *f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);
    if (strcmp(type, "P2") == 0)
        numChannels = 1;
    else if (strcmp(type, "P3") == 0)
        numChannels = 3;
    else // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    pixels = (uint8_t *)malloc(width * height * numChannels * sizeof(uint8_t));
    for (int i = 0; i < width * height * numChannels; i++)
        fscanf(f, "%hhu", &pixels[i]);

    fclose(f);
}

void writePnm(uint8_t *pixels, int numChannels, int width, int height, char *fileName)
{
    FILE *f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    if (numChannels == 1)
        fprintf(f, "P2\n%i %i\n255\n", width, height);
    else if (numChannels == 3)
        fprintf(f, "P3\n%i %i\n255\n", width, height);
    else
    {
        fclose(f);
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }
    for (int h = 0; h < height * numChannels; h++)
    {
        for (int w = 0; w < width; w++)
        {
            fprintf(f, "%hhu ", pixels[h * width + w]);
            if (w == width - 1)
            {
                fprintf(f, "\n");
            }
        }
    }
    fclose(f);
}

// ======================================== KERNELs ========================================
__global__ void grayscaleOnDevice(int width, int height, uint8_t *inPixels, uint8_t *grayscalePixels)
{
    // for (int r = 0; r < height; r++)
    // {
    //     for (int c = 0; c < width; c++)
    //     {
    //         int i = r * width + c;
    //         uint8_t red = inPixels[3 * i];
    //         uint8_t green = inPixels[3 * i + 1];
    //         uint8_t blue = inPixels[3 * i + 2];
    //         grayscalePixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
    //     }
    // }
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        int i = r * width + c;
        uint8_t red = inPixels[3 * i];
        uint8_t green = inPixels[3 * i + 1];
        uint8_t blue = inPixels[3 * i + 2];
        grayscalePixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
    }
}

__global__ void convolutionOnDevice(int width, int height, int filterWidth, uint8_t *grayscalePixels, float *filter1, float *filter2, uint8_t *convoPixels)
{
    // for (int r = 0; r < height; r++)
    // {
    //     for (int c = 0; c < width; c++)
    //     {
    //         float new_pixel_1 = 0;
    //         float new_pixel_2 = 0;
    //         for (int j = 0; j < filterWidth; j++)
    //         {
    //             for (int k = 0; k < filterWidth; k++)
    //             {
    //                 int current_row = r + j - (filterWidth / 2);
    //                 int current_column = c + k - (filterWidth / 2);
    //                 if (current_row < 0)
    //                 {
    //                     current_row = 0;
    //                 }
    //                 else if (current_row > height - 1)
    //                 {
    //                     current_row = height - 1;
    //                 }
    //                 if (current_column < 0)
    //                 {
    //                     current_column = 0;
    //                 }
    //                 else if (current_column > width - 1)
    //                 {
    //                     current_column = width - 1;
    //                 }
    //                 int i = current_row * width + current_column;
    //                 new_pixel_1 += grayscalePixels[i] * filter1[j * filterWidth + k];
    //                 new_pixel_2 += grayscalePixels[i] * filter2[j * filterWidth + k];
    //             }
    //         }
    //         float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
    //         if (new_pixel > 255)
    //         {
    //             new_pixel = 255;
    //         }
    //         else if (new_pixel < 0)
    //         {
    //             new_pixel = 0;
    //         }
    //         int i = r * width + c;
    //         convoPixels[i] = new_pixel;
    //     }
    // }

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    // // For dc_filter debug
    // if (r == 0 && c == 0)
    // {
    //     for (int i = 0; i < filterWidth; i++)
    //     {
    //         for (int j = 0; j < filterWidth; j++)
    //         {
    //             printf("%.0f %.0f | ", filter2[i * filterWidth + j], dc_filter2[i * filterWidth + j]);
    //         }
    //     }
    // }

    if (r < height && c < width)
    {
        float new_pixel_1 = 0;
        float new_pixel_2 = 0;
        for (int j = 0; j < filterWidth; j++)
        {
            for (int k = 0; k < filterWidth; k++)
            {
                int current_row = r + j - (filterWidth / 2);
                int current_column = c + k - (filterWidth / 2);
                if (current_row < 0)
                {
                    current_row = 0;
                }
                else if (current_row > height - 1)
                {
                    current_row = height - 1;
                }
                if (current_column < 0)
                {
                    current_column = 0;
                }
                else if (current_column > width - 1)
                {
                    current_column = width - 1;
                }
                int i = current_row * width + current_column;
                new_pixel_1 += grayscalePixels[i] * filter1[j * filterWidth + k];
                new_pixel_2 += grayscalePixels[i] * filter2[j * filterWidth + k];
            }
        }
        float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
        if (new_pixel > 255)
        {
            new_pixel = 255;
        }
        else if (new_pixel < 0)
        {
            new_pixel = 0;
        }
        int i = r * width + c;
        convoPixels[i] = new_pixel;
    }
}

__global__ void convolutionOnDeviceOpt1(int width, int height, int filterWidth, uint8_t *grayscalePixels, uint8_t *convoPixels)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        float new_pixel_1 = 0;
        float new_pixel_2 = 0;
        for (int j = 0; j < filterWidth; j++)
        {
            for (int k = 0; k < filterWidth; k++)
            {
                int current_row = r + j - (filterWidth / 2);
                int current_column = c + k - (filterWidth / 2);
                if (current_row < 0)
                {
                    current_row = 0;
                }
                else if (current_row > height - 1)
                {
                    current_row = height - 1;
                }
                if (current_column < 0)
                {
                    current_column = 0;
                }
                else if (current_column > width - 1)
                {
                    current_column = width - 1;
                }
                int i = current_row * width + current_column;
                new_pixel_1 += grayscalePixels[i] * dc_filter1[j * filterWidth + k];
                new_pixel_2 += grayscalePixels[i] * dc_filter2[j * filterWidth + k];
            }
        }
        float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
        if (new_pixel > 255)
        {
            new_pixel = 255;
        }
        else if (new_pixel < 0)
        {
            new_pixel = 0;
        }
        int i = r * width + c;
        convoPixels[i] = new_pixel;
    }
}

__global__ void minEnergiesOnDevice(int width, int height, uint8_t *convoPixels, uint8_t *backtrack, uint8_t *minEnergies)
{
    // // The remained above rows
    // for (int r = 1; r < height; r++)
    // {
    //     for (int c = 0; c < width; c++)
    //     {
    //         int i = (height - 1 - r) * width + c;
    //         int e[3] = {99999, 99999, 99999};
    //         if (c == 0)
    //         {
    //             e[1] = minEnergies[((height - 1 - r) + 1) * width + c];
    //             e[2] = minEnergies[((height - 1 - r) + 1) * width + (c + 1)];
    //         }
    //         else if (c == width - 1)
    //         {
    //             e[0] = minEnergies[((height - 1 - r) + 1) * width + (c - 1)];
    //             e[1] = minEnergies[((height - 1 - r) + 1) * width + c];
    //         }
    //         else
    //         {
    //             e[0] = minEnergies[((height - 1 - r) + 1) * width + (c - 1)];
    //             e[1] = minEnergies[((height - 1 - r) + 1) * width + c];
    //             e[2] = minEnergies[((height - 1 - r) + 1) * width + (c + 1)];
    //         }
    //         // uint8_t min_idx = min_e_idx(e[0], e[1], e[2]);
    //         uint8_t min_idx = 255;
    //         if (e[1] <= e[0] && e[1] <= e[2])
    //         {
    //             min_idx = 1; // return 1;
    //         }
    //         else if (e[2] <= e[0] && e[2] <= e[1])
    //         {
    //             min_idx = 2; // return 2;
    //         }
    //         else if (e[0] <= e[1] && e[0] <= e[2])
    //         {
    //             min_idx = 0; // return 0;
    //         }
    //         int tempMinE = convoPixels[i] + e[min_idx]; // Calculate 1 minimal energy base on 3-energies-below
    //         backtrack[i] = min_idx;                     // Save the direction (which of the 3-energies-below)
    //         if (tempMinE > 255)
    //         {
    //             tempMinE = 255;
    //         }
    //         else if (tempMinE < 0)
    //         {
    //             tempMinE = 0;
    //         }
    //         minEnergies[i] = tempMinE * 0.9; // Special
    //     }
    // }

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        // The remained above rows
        if (r != 0)
        {
        }
    }
}

__global__ void vMinEnergiesOnDevice(int width, int height, uint8_t *convoPixels, int *vMinEnergies)
{

    // int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (c < width)
    {
        int temp_sum = 0;
        for (int r = 0; r < height; r++)
        {
            int i = r * width + c;
            temp_sum += convoPixels[i];
        }
        vMinEnergies[c] = temp_sum;
    }
}

__global__ void minSeamBacktrackOnDevice(int width, int height, uint8_t *backtrack, int *min_seam_track)
{
    // for (int r = 1; r < height; r++)
    // {
    //     int prev_c = min_seam_track[r - 1];
    //     uint8_t direction = backtrack[(r - 1) * width + prev_c];
    //     if (direction == 0)
    //     {
    //         if (prev_c == 0)
    //         {
    //             min_seam_track[r] = 0;
    //         }
    //         else
    //         {
    //             min_seam_track[r] = prev_c - 1;
    //         }
    //     }
    //     else if (direction == 1)
    //     {
    //         min_seam_track[r] = prev_c;
    //     }
    //     else if (direction == 2)
    //     {
    //         if (prev_c == width - 1)
    //         {
    //             min_seam_track[r] = width - 1;
    //         }
    //         else
    //         {
    //             min_seam_track[r] = prev_c + 1;
    //         }
    //     }
    // }

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
    }
}

__global__ void removeMinSeamOnDevice(int width, int height, uint8_t *inPixels, int *min_seam_track, uint8_t *outPixels)
{
    // for (int r = 0; r < height; r++)
    // {
    //     int ignored_idx = min_seam_track[r];
    //     bool meet_ignored_pixel_yet = false;
    //     for (int c = 0; c < width; c++)
    //     {
    //         int i = r * width + c;
    //         if (meet_ignored_pixel_yet == false)
    //         {
    //             if (c != ignored_idx)
    //             {
    //                 outPixels[i * 3 + 0 - 3 * r] = inPixels[i * 3 + 0];
    //                 outPixels[i * 3 + 1 - 3 * r] = inPixels[i * 3 + 1];
    //                 outPixels[i * 3 + 2 - 3 * r] = inPixels[i * 3 + 2];
    //             }
    //             else
    //             {
    //                 meet_ignored_pixel_yet = true;
    //             }
    //         }
    //         else
    //         {
    //             outPixels[i * 3 + 0 - 3 * r - 3] = inPixels[i * 3 + 0];
    //             outPixels[i * 3 + 1 - 3 * r - 3] = inPixels[i * 3 + 1];
    //             outPixels[i * 3 + 2 - 3 * r - 3] = inPixels[i * 3 + 2];
    //         }
    //     }
    // }

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    // int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height)
    {
        int ignored_idx = min_seam_track[r];
        bool meet_ignored_pixel_yet = false;
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            if (meet_ignored_pixel_yet == false)
            {
                if (c != ignored_idx)
                {
                    outPixels[i * 3 + 0 - 3 * r] = inPixels[i * 3 + 0];
                    outPixels[i * 3 + 1 - 3 * r] = inPixels[i * 3 + 1];
                    outPixels[i * 3 + 2 - 3 * r] = inPixels[i * 3 + 2];
                }
                else
                {
                    meet_ignored_pixel_yet = true;
                }
            }
            else
            {
                outPixels[i * 3 + 0 - 3 * r - 3] = inPixels[i * 3 + 0];
                outPixels[i * 3 + 1 - 3 * r - 3] = inPixels[i * 3 + 1];
                outPixels[i * 3 + 2 - 3 * r - 3] = inPixels[i * 3 + 2];
            }
        }
    }
}

// ======================================== HOST ========================================
void seamCarvingOnHost(uint8_t *inPixels, char *fileName, int numChannels, int width, int height, float *filter1, float *filter2, int filterWidth, dim3 blockSize, uint8_t *grayscalePixels, uint8_t *convoPixels, uint8_t *minEnergies, uint8_t *backtrack, int *vMinEnergies, int *min_seam_track, uint8_t *outPixels, int *minEnergiesInt)
{
    // ============================== Grayscale ==============================

    // uint8_t *grayscalePixels = (uint8_t *)malloc(width * height * 1);
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            uint8_t red = inPixels[3 * i];
            uint8_t green = inPixels[3 * i + 1];
            uint8_t blue = inPixels[3 * i + 2];
            grayscalePixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
        }
    }
    writePnm(grayscalePixels, 1, width, height, concatStr("out_gray", ".pnm")); // Just to test by eyes

    // ============================== Convolution ==============================

    // uint8_t *convoPixels = (uint8_t *)malloc(width * height * 1);
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            float new_pixel_1 = 0;
            float new_pixel_2 = 0;
            for (int j = 0; j < filterWidth; j++)
            {
                for (int k = 0; k < filterWidth; k++)
                {
                    int current_row = r + j - (filterWidth / 2);
                    int current_column = c + k - (filterWidth / 2);
                    if (current_row < 0)
                    {
                        current_row = 0;
                    }
                    else if (current_row > height - 1)
                    {
                        current_row = height - 1;
                    }
                    if (current_column < 0)
                    {
                        current_column = 0;
                    }
                    else if (current_column > width - 1)
                    {
                        current_column = width - 1;
                    }
                    int i = current_row * width + current_column;
                    new_pixel_1 += grayscalePixels[i] * filter1[j * filterWidth + k];
                    new_pixel_2 += grayscalePixels[i] * filter2[j * filterWidth + k];
                }
            }
            float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
            if (new_pixel > 255)
            {
                new_pixel = 255;
            }
            else if (new_pixel < 0)
            {
                new_pixel = 0;
            }
            int i = r * width + c;
            convoPixels[i] = new_pixel;
        }
    }
    writePnm(convoPixels, 1, width, height, concatStr("out_edge", ".pnm")); // Just to test by eyes

    // ============================== Energy to Min Energy ==============================

    // Loop from bottom to top row
    // -> Calculate the min energy of each pixel:
    //    +) The bottom row: Just copy the energy -> min energy
    //    +) The remained above rows: Calculate min energy on the current row by using:
    //                                -) The current energy
    //                                -) The min of 3-min-energy-on-the-row-below
    // Simultaneously backtrack the direction of each pixel (which of the 3-energies-below)

    // uint8_t *minEnergies = (uint8_t *)malloc(width * height * 1);
    // uint8_t *backtrack = (uint8_t *)malloc(width * height * 1);
    for (int r = height - 1; r >= 0; r--)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int tempMinE;

            // The bottom row
            if (r == height - 1)
            {
                tempMinE = convoPixels[i];
                backtrack[i] = 1;
            }
            // The remained above rows
            else
            {
                int e[3] = {99999, 99999, 99999};
                if (c == 0)
                {
                    e[1] = minEnergiesInt[(r + 1) * width + c];
                    e[2] = minEnergiesInt[(r + 1) * width + (c + 1)];
                }
                else if (c == width - 1)
                {
                    e[0] = minEnergiesInt[(r + 1) * width + (c - 1)];
                    e[1] = minEnergiesInt[(r + 1) * width + c];
                }
                else
                {
                    e[0] = minEnergiesInt[(r + 1) * width + (c - 1)];
                    e[1] = minEnergiesInt[(r + 1) * width + c];
                    e[2] = minEnergiesInt[(r + 1) * width + (c + 1)];
                }
                uint8_t min_idx = min_e_idx(e[0], e[1], e[2]);

                tempMinE = convoPixels[i] + e[min_idx]; // Calculate 1 minimal energy base on 3-energies-below
                backtrack[i] = min_idx;                 // Save the direction (which of the 3-energies-below)
            }

            // if (tempMinE > 255)
            // {
            //     tempMinE = 255;
            // }
            // else if (tempMinE < 0)
            // {
            //     tempMinE = 0;
            // }
            // minEnergies[i] = tempMinE * 0.9; // Special

            minEnergiesInt[i] = tempMinE * 1.0;
        }
    }
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int tempMinE = 0;
            tempMinE = minEnergiesInt[i];
            minEnergies[i] = tempMinE * 0.02;
        }
    }
    writePnm(minEnergies, 1, width, height, concatStr("out_minEnergy", ".pnm")); // Just to test by eyes

    // ============================== Extra: Vertical Min Energy ==============================

    // Loop through columns and calculate the sum of energies of each column
    // This will be used to help to choose which pixel to start when find the min seam later

    // int *vMinEnergies = (int *)malloc(width);
    for (int c = 0; c < width; c++)
    {
        int temp_sum = 0;
        for (int r = 0; r < height; r++)
        {
            int i = r * width + c;
            temp_sum += convoPixels[i];
        }
        vMinEnergies[c] = temp_sum;
    }

    // ============================== Find the Min Seam ==============================

    // First: Find the best pixel (on the top row) to start.
    //        This pixel will be used to start the backtrack.
    // Finding that starting pixel will depend on:
    // +) The min energy of that pixel (of course, the min value on the row)
    // +) If there are multiple pixels have same min value
    //    -> Use vMinEnergies calculated above

    // int *min_seam_track = (int *)malloc(height);
    for (int c = 0; c < width; c++)
    {
        uint8_t temp_min_e;
        int temp_min_v_e;
        if (c == 0)
        {
            temp_min_e = minEnergies[0];
            temp_min_v_e = vMinEnergies[0];
            min_seam_track[0] = 0;
        }
        else
        {
            bool is_satisfied = false;
            // ----- Just a trick -----
            if (width % 2 == 0)
            {
                if (minEnergies[c] < temp_min_e && vMinEnergies[c] <= temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            else
            {
                if (minEnergies[c] <= temp_min_e && vMinEnergies[c] < temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            if (is_satisfied == true)
            {
                temp_min_e = minEnergies[c];
                temp_min_v_e = vMinEnergies[c];
                min_seam_track[0] = c;
            }
        }
    }

    // Second: With the starting pixel:
    //         Backtrack from row to row to find a list of pixels
    //         -> The seam with min energy

    for (int r = 1; r < height; r++)
    {
        int prev_c = min_seam_track[r - 1];
        uint8_t direction = backtrack[(r - 1) * width + prev_c];
        if (direction == 0)
        {
            if (prev_c == 0)
            {
                min_seam_track[r] = 0;
            }
            else
            {
                min_seam_track[r] = prev_c - 1;
            }
        }
        else if (direction == 1)
        {
            min_seam_track[r] = prev_c;
        }
        else if (direction == 2)
        {
            if (prev_c == width - 1)
            {
                min_seam_track[r] = width - 1;
            }
            else
            {
                min_seam_track[r] = prev_c + 1;
            }
        }
    }

    // ============================== Remove Min Seam from image ==============================

    // Copy all pixels from inPixels to outPixels, EXCEPT the min seam
    // Just it!

    // uint8_t *outPixels = (uint8_t *)malloc((width - 1) * height * numChannels * sizeof(uint8_t)); // width-1: means that the output image will be cut 1 pixel (the min seam)
    for (int r = 0; r < height; r++)
    {
        int ignored_idx = min_seam_track[r];
        bool meet_ignored_pixel_yet = false;
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            if (meet_ignored_pixel_yet == false)
            {
                if (c != ignored_idx)
                {
                    outPixels[i * 3 + 0 - 3 * r] = inPixels[i * 3 + 0];
                    outPixels[i * 3 + 1 - 3 * r] = inPixels[i * 3 + 1];
                    outPixels[i * 3 + 2 - 3 * r] = inPixels[i * 3 + 2];
                }
                else
                {
                    meet_ignored_pixel_yet = true;
                }
            }
            else
            {
                outPixels[i * 3 + 0 - 3 * r - 3] = inPixels[i * 3 + 0];
                outPixels[i * 3 + 1 - 3 * r - 3] = inPixels[i * 3 + 1];
                outPixels[i * 3 + 2 - 3 * r - 3] = inPixels[i * 3 + 2];
            }
        }
    }
    writePnm(outPixels, 3, width - 1, height, fileName); // Just to test by eyes
    // ============================== =========================== ==============================
}

// ======================================== DEVICE ========================================
void seamCarvingOnDevice(uint8_t *inPixels, char *fileName, int numChannels, int width, int height, float *filter1, float *filter2, int filterWidth, dim3 blockSize, uint8_t *grayscalePixels, uint8_t *convoPixels, uint8_t *minEnergies, uint8_t *backtrack, int *vMinEnergies, int *min_seam_track, uint8_t *outPixels, uint8_t *d_inPixels, size_t nBytes_inPixels, uint8_t *d_grayscalePixels, size_t nBytes_grayscalePixels, uint8_t *d_convoPixels, size_t nBytes_convoPixels, float *d_filter1, float *d_filter2, size_t nBytes_filter, uint8_t *d_backtrack, size_t nBytes_backtrack, uint8_t *d_minEnergies, size_t nBytes_minEnergies, int *d_vMinEnergies, size_t nBytes_vMinEnergies, int *d_min_seam_track, size_t nBytes_min_seam_track, uint8_t *d_outPixels, size_t nBytes_outPixels, int *minEnergiesInt)
{
    // Gridsize
    dim3 gridSize(
        (width - 1) / blockSize.x + 1,
        (height - 1) / blockSize.y + 1);

    // ============================== Grayscale (Parallel) ==============================

    // uint8_t *grayscalePixels = (uint8_t *)malloc(width * height * 1);
    // for (int r = 0; r < height; r++)
    // {
    //     for (int c = 0; c < width; c++)
    //     {
    //         int i = r * width + c;
    //         uint8_t red = inPixels[3 * i];
    //         uint8_t green = inPixels[3 * i + 1];
    //         uint8_t blue = inPixels[3 * i + 2];
    //         grayscalePixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
    //     }
    // }

    // Copy data
    CHECK(hipMemcpy(d_inPixels, inPixels, nBytes_inPixels, hipMemcpyHostToDevice));
    // Call kernel
    grayscaleOnDevice<<<gridSize, blockSize>>>(width, height, d_inPixels, d_grayscalePixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(grayscalePixels, d_grayscalePixels, nBytes_grayscalePixels, hipMemcpyDeviceToHost));

    // Output
    writePnm(grayscalePixels, 1, width, height, concatStr("out_gray", ".pnm"));

    // ============================== Convolution (Parallel) ==============================

    // uint8_t *convoPixels = (uint8_t *)malloc(width * height * 1);
    // for (int r = 0; r < height; r++)
    // {
    //     for (int c = 0; c < width; c++)
    //     {
    //         float new_pixel_1 = 0;
    //         float new_pixel_2 = 0;
    //         for (int j = 0; j < filterWidth; j++)
    //         {
    //             for (int k = 0; k < filterWidth; k++)
    //             {
    //                 int current_row = r + j - (filterWidth / 2);
    //                 int current_column = c + k - (filterWidth / 2);
    //                 if (current_row < 0)
    //                 {
    //                     current_row = 0;
    //                 }
    //                 else if (current_row > height - 1)
    //                 {
    //                     current_row = height - 1;
    //                 }
    //                 if (current_column < 0)
    //                 {
    //                     current_column = 0;
    //                 }
    //                 else if (current_column > width - 1)
    //                 {
    //                     current_column = width - 1;
    //                 }
    //                 int i = current_row * width + current_column;
    //                 new_pixel_1 += grayscalePixels[i] * filter1[j * filterWidth + k];
    //                 new_pixel_2 += grayscalePixels[i] * filter2[j * filterWidth + k];
    //             }
    //         }
    //         float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
    //         if (new_pixel > 255)
    //         {
    //             new_pixel = 255;
    //         }
    //         else if (new_pixel < 0)
    //         {
    //             new_pixel = 0;
    //         }
    //         int i = r * width + c;
    //         convoPixels[i] = new_pixel;
    //     }
    // }

    // Copy data
    CHECK(hipMemcpy(d_grayscalePixels, grayscalePixels, nBytes_grayscalePixels, hipMemcpyHostToDevice)); // No need
    CHECK(hipMemcpy(d_filter1, filter1, nBytes_filter, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_filter2, filter2, nBytes_filter, hipMemcpyHostToDevice));
    // Call kernel
    convolutionOnDevice<<<gridSize, blockSize>>>(width, height, filterWidth, d_grayscalePixels, d_filter1, d_filter2, d_convoPixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(convoPixels, d_convoPixels, nBytes_convoPixels, hipMemcpyDeviceToHost));

    // Output
    writePnm(convoPixels, 1, width, height, concatStr("out_edge", ".pnm"));

    // ============================== Energy to Min Energy ==============================

    // Loop from bottom to top row
    // -> Calculate the min energy of each pixel:
    //    +) The bottom row: Just copy the energy -> min energy
    //    +) The remained above rows: Calculate min energy on the current row by using:
    //                                -) The current energy
    //                                -) The min of 3-min-energy-on-the-row-below
    // Simultaneously backtrack the direction of each pixel (which of the 3-energies-below)

    // uint8_t *minEnergies = (uint8_t *)malloc(width * height * 1);
    // uint8_t *backtrack = (uint8_t *)malloc(width * height * 1);

    // // The bottom row
    for (int c = 0; c < width; c++)
    {
        int i = (height - 1) * width + c;
        int tempMinE = convoPixels[i];

        // if (tempMinE > 255)
        // {
        //     tempMinE = 255;
        // }
        // else if (tempMinE < 0)
        // {
        //     tempMinE = 0;
        // }
        // minEnergies[i] = tempMinE * 0.9; // Special

        minEnergiesInt[i] = tempMinE * 1.0;

        backtrack[i] = 1;
    }

    // // The remained above rows
    for (int r = 1; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = (height - 1 - r) * width + c;
            int e[3] = {99999, 99999, 99999};
            if (c == 0)
            {
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            else if (c == width - 1)
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
            }
            else
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            // uint8_t min_idx = min_e_idx(e[0], e[1], e[2]);
            uint8_t min_idx = 255;
            if (e[1] <= e[0] && e[1] <= e[2])
            {
                min_idx = 1; // return 1;
            }
            else if (e[2] <= e[0] && e[2] <= e[1])
            {
                min_idx = 2; // return 2;
            }
            else if (e[0] <= e[1] && e[0] <= e[2])
            {
                min_idx = 0; // return 0;
            }
            int tempMinE = convoPixels[i] + e[min_idx]; // Calculate 1 minimal energy base on 3-energies-below

            // if (tempMinE > 255)
            // {
            //     tempMinE = 255;
            // }
            // else if (tempMinE < 0)
            // {
            //     tempMinE = 0;
            // }
            // minEnergies[i] = tempMinE * 0.9; // Special

            minEnergiesInt[i] = tempMinE * 1.0;

            backtrack[i] = min_idx; // Save the direction (which of the 3-energies-below)
        }
    }

    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int tempMinE = 0;
            tempMinE = minEnergiesInt[i];
            minEnergies[i] = tempMinE * 0.02;
        }
    }

    // Output
    writePnm(minEnergies, 1, width, height, concatStr("out_minEnergy", ".pnm"));

    // ============================== Extra: Vertical Min Energy (Parallel) ==============================

    // Loop through columns and calculate the sum of energies of each column
    // This will be used to help to choose which pixel to start when find the min seam later

    // int *vMinEnergies = (int *)malloc(width);
    // for (int c = 0; c < width; c++)
    // {
    //     int temp_sum = 0;
    //     for (int r = 0; r < height; r++)
    //     {
    //         int i = r * width + c;
    //         temp_sum += convoPixels[i];
    //     }
    //     vMinEnergies[c] = temp_sum;
    // }

    // Copy data
    CHECK(hipMemcpy(d_convoPixels, convoPixels, nBytes_convoPixels, hipMemcpyHostToDevice)); // No need
    // Call kernel
    vMinEnergiesOnDevice<<<gridSize, blockSize>>>(width, height, d_convoPixels, d_vMinEnergies);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(vMinEnergies, d_vMinEnergies, nBytes_vMinEnergies, hipMemcpyDeviceToHost));

    // ============================== Find the Min Seam ==============================

    // ------------------------------ First step (No parallel) ------------------------------

    // First: Find the best pixel (on the top row) to start.
    //        This pixel will be used to start the backtrack.
    // Finding that starting pixel will depend on:
    // +) The min energy of that pixel (of course, the min value on the row)
    // +) If there are multiple pixels have same min value
    //    -> Use vMinEnergies calculated above

    // int *min_seam_track = (int *)malloc(height);
    for (int c = 0; c < width; c++)
    {
        uint8_t temp_min_e;
        int temp_min_v_e;
        if (c == 0)
        {
            temp_min_e = minEnergies[0];
            temp_min_v_e = vMinEnergies[0];
            min_seam_track[0] = 0;
        }
        else
        {
            bool is_satisfied = false;
            // ----- Just a trick -----
            if (width % 2 == 0)
            {
                if (minEnergies[c] < temp_min_e && vMinEnergies[c] <= temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            else
            {
                if (minEnergies[c] <= temp_min_e && vMinEnergies[c] < temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            if (is_satisfied == true)
            {
                temp_min_e = minEnergies[c];
                temp_min_v_e = vMinEnergies[c];
                min_seam_track[0] = c;
            }
        }
    }

    // ------------------------------ Second step (No parallel) ------------------------------

    // Second: With the starting pixel:
    //         Backtrack from row to row to find a list of pixels
    //         -> The seam with min energy

    for (int r = 1; r < height; r++)
    {
        int prev_c = min_seam_track[r - 1];
        uint8_t direction = backtrack[(r - 1) * width + prev_c];
        if (direction == 0)
        {
            if (prev_c == 0)
            {
                min_seam_track[r] = 0;
            }
            else
            {
                min_seam_track[r] = prev_c - 1;
            }
        }
        else if (direction == 1)
        {
            min_seam_track[r] = prev_c;
        }
        else if (direction == 2)
        {
            if (prev_c == width - 1)
            {
                min_seam_track[r] = width - 1;
            }
            else
            {
                min_seam_track[r] = prev_c + 1;
            }
        }
    }

    // ============================== Remove Min Seam from image (Parallel) ==============================

    // Copy all pixels from inPixels to outPixels, EXCEPT the min seam
    // Just it!

    // uint8_t *outPixels = (uint8_t *)malloc((width - 1) * height * numChannels * sizeof(uint8_t)); // width-1: means that the output image will be cut 1 pixel (the min seam)
    // for (int r = 0; r < height; r++)
    // {
    //     int ignored_idx = min_seam_track[r];
    //     bool meet_ignored_pixel_yet = false;
    //     for (int c = 0; c < width; c++)
    //     {
    //         int i = r * width + c;
    //         if (meet_ignored_pixel_yet == false)
    //         {
    //             if (c != ignored_idx)
    //             {
    //                 outPixels[i * 3 + 0 - 3 * r] = inPixels[i * 3 + 0];
    //                 outPixels[i * 3 + 1 - 3 * r] = inPixels[i * 3 + 1];
    //                 outPixels[i * 3 + 2 - 3 * r] = inPixels[i * 3 + 2];
    //             }
    //             else
    //             {
    //                 meet_ignored_pixel_yet = true;
    //             }
    //         }
    //         else
    //         {
    //             outPixels[i * 3 + 0 - 3 * r - 3] = inPixels[i * 3 + 0];
    //             outPixels[i * 3 + 1 - 3 * r - 3] = inPixels[i * 3 + 1];
    //             outPixels[i * 3 + 2 - 3 * r - 3] = inPixels[i * 3 + 2];
    //         }
    //     }
    // }

    // Copy data
    CHECK(hipMemcpy(d_inPixels, inPixels, nBytes_inPixels, hipMemcpyHostToDevice)); // No need
    CHECK(hipMemcpy(d_min_seam_track, min_seam_track, nBytes_min_seam_track, hipMemcpyHostToDevice));
    // Call kernel
    removeMinSeamOnDevice<<<gridSize, blockSize>>>(width, height, d_inPixels, d_min_seam_track, d_outPixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(outPixels, d_outPixels, nBytes_outPixels, hipMemcpyDeviceToHost));

    // Output
    writePnm(outPixels, 3, width - 1, height, fileName);
    // ============================== =========================== ==============================
}

// ======================================== DEVICE (OPTIMIZED) ========================================
void seamCarvingOnDeviceOpt1(uint8_t *inPixels, char *fileName, int numChannels, int width, int height, int filterWidth, dim3 blockSize, uint8_t *grayscalePixels, uint8_t *convoPixels, uint8_t *minEnergies, uint8_t *backtrack, int *vMinEnergies, int *min_seam_track, uint8_t *outPixels, uint8_t *d_inPixels, size_t nBytes_inPixels, uint8_t *d_grayscalePixels, size_t nBytes_grayscalePixels, uint8_t *d_convoPixels, size_t nBytes_convoPixels, uint8_t *d_backtrack, size_t nBytes_backtrack, uint8_t *d_minEnergies, size_t nBytes_minEnergies, int *d_vMinEnergies, size_t nBytes_vMinEnergies, int *d_min_seam_track, size_t nBytes_min_seam_track, uint8_t *d_outPixels, size_t nBytes_outPixels, int *minEnergiesInt)
{
    // Gridsize
    dim3 gridSize(
        (width - 1) / blockSize.x + 1,
        (height - 1) / blockSize.y + 1);

    // ============================== Grayscale (Parallel) ==============================

    // Copy data
    CHECK(hipMemcpy(d_inPixels, inPixels, nBytes_inPixels, hipMemcpyHostToDevice));
    // Call kernel
    grayscaleOnDevice<<<gridSize, blockSize>>>(width, height, d_inPixels, d_grayscalePixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(grayscalePixels, d_grayscalePixels, nBytes_grayscalePixels, hipMemcpyDeviceToHost));

    // // Output
    // writePnm(grayscalePixels, 1, width, height, concatStr("out_gray", ".pnm"));

    // ============================== Convolution (Parallel) ==============================

    // Copy data
    // CHECK(cudaMemcpy(d_grayscalePixels, grayscalePixels, nBytes_grayscalePixels, cudaMemcpyHostToDevice)); // No need
    // CHECK(cudaMemcpy(d_filter1, filter1, nBytes_filter, cudaMemcpyHostToDevice));
    // CHECK(cudaMemcpy(d_filter2, filter2, nBytes_filter, cudaMemcpyHostToDevice));
    // Call kernel
    convolutionOnDeviceOpt1<<<gridSize, blockSize>>>(width, height, filterWidth, d_grayscalePixels, d_convoPixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(convoPixels, d_convoPixels, nBytes_convoPixels, hipMemcpyDeviceToHost));

    // // Output
    // writePnm(convoPixels, 1, width, height, concatStr("out_edge", ".pnm"));

    // ============================== Energy to Min Energy ==============================

    // Loop from bottom to top row
    // -> Calculate the min energy of each pixel:
    //    +) The bottom row: Just copy the energy -> min energy
    //    +) The remained above rows: Calculate min energy on the current row by using:
    //                                -) The current energy
    //                                -) The min of 3-min-energy-on-the-row-below
    // Simultaneously backtrack the direction of each pixel (which of the 3-energies-below)

    // // The bottom row
    for (int c = 0; c < width; c++)
    {
        int i = (height - 1) * width + c;
        int tempMinE = convoPixels[i];
        minEnergiesInt[i] = tempMinE * 1.0;
        backtrack[i] = 1;
    }

    // // The remained above rows
    for (int r = 1; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = (height - 1 - r) * width + c;
            int e[3] = {99999, 99999, 99999};
            if (c == 0)
            {
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            else if (c == width - 1)
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
            }
            else
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            // uint8_t min_idx = min_e_idx(e[0], e[1], e[2]);
            uint8_t min_idx = 255;
            if (e[1] <= e[0] && e[1] <= e[2])
            {
                min_idx = 1; // return 1;
            }
            else if (e[2] <= e[0] && e[2] <= e[1])
            {
                min_idx = 2; // return 2;
            }
            else if (e[0] <= e[1] && e[0] <= e[2])
            {
                min_idx = 0; // return 0;
            }
            int tempMinE = convoPixels[i] + e[min_idx]; // Calculate 1 minimal energy base on 3-energies-below
            minEnergiesInt[i] = tempMinE * 1.0;
            backtrack[i] = min_idx; // Save the direction (which of the 3-energies-below)
        }
    }

    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int tempMinE = 0;
            tempMinE = minEnergiesInt[i];
            minEnergies[i] = tempMinE * 0.02;
        }
    }

    // // Output
    // writePnm(minEnergies, 1, width, height, concatStr("out_minEnergy", ".pnm"));
    // ============================== Extra: Vertical Min Energy (Parallel) ==============================

    // Loop through columns and calculate the sum of energies of each column
    // This will be used to help to choose which pixel to start when find the min seam later

    // Copy data
    // CHECK(cudaMemcpy(d_convoPixels, convoPixels, nBytes_convoPixels, cudaMemcpyHostToDevice)); // No need
    // Call kernel
    vMinEnergiesOnDevice<<<gridSize, blockSize>>>(width, height, d_convoPixels, d_vMinEnergies);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(vMinEnergies, d_vMinEnergies, nBytes_vMinEnergies, hipMemcpyDeviceToHost));

    // ============================== Find the Min Seam ==============================

    // ------------------------------ First step (No parallel) ------------------------------

    // First: Find the best pixel (on the top row) to start.
    //        This pixel will be used to start the backtrack.
    // Finding that starting pixel will depend on:
    // +) The min energy of that pixel (of course, the min value on the row)
    // +) If there are multiple pixels have same min value
    //    -> Use vMinEnergies calculated above

    for (int c = 0; c < width; c++)
    {
        uint8_t temp_min_e;
        int temp_min_v_e;
        if (c == 0)
        {
            temp_min_e = minEnergies[0];
            temp_min_v_e = vMinEnergies[0];
            min_seam_track[0] = 0;
        }
        else
        {
            bool is_satisfied = false;
            // ----- Just a trick -----
            if (width % 2 == 0)
            {
                if (minEnergies[c] < temp_min_e && vMinEnergies[c] <= temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            else
            {
                if (minEnergies[c] <= temp_min_e && vMinEnergies[c] < temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            if (is_satisfied == true)
            {
                temp_min_e = minEnergies[c];
                temp_min_v_e = vMinEnergies[c];
                min_seam_track[0] = c;
            }
        }
    }

    // ------------------------------ Second step (No parallel) ------------------------------

    // Second: With the starting pixel:
    //         Backtrack from row to row to find a list of pixels
    //         -> The seam with min energy

    for (int r = 1; r < height; r++)
    {
        int prev_c = min_seam_track[r - 1];
        uint8_t direction = backtrack[(r - 1) * width + prev_c];
        if (direction == 0)
        {
            if (prev_c == 0)
            {
                min_seam_track[r] = 0;
            }
            else
            {
                min_seam_track[r] = prev_c - 1;
            }
        }
        else if (direction == 1)
        {
            min_seam_track[r] = prev_c;
        }
        else if (direction == 2)
        {
            if (prev_c == width - 1)
            {
                min_seam_track[r] = width - 1;
            }
            else
            {
                min_seam_track[r] = prev_c + 1;
            }
        }
    }

    // ============================== Remove Min Seam from image (Parallel) ==============================

    // Copy all pixels from inPixels to outPixels, EXCEPT the min seam
    // Just it!

    // Copy data
    // CHECK(cudaMemcpy(d_inPixels, inPixels, nBytes_inPixels, cudaMemcpyHostToDevice)); // No need
    CHECK(hipMemcpy(d_min_seam_track, min_seam_track, nBytes_min_seam_track, hipMemcpyHostToDevice));
    // Call kernel
    removeMinSeamOnDevice<<<gridSize, blockSize>>>(width, height, d_inPixels, d_min_seam_track, d_outPixels);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(outPixels, d_outPixels, nBytes_outPixels, hipMemcpyDeviceToHost));

    // Output
    writePnm(outPixels, 3, width - 1, height, fileName);
    // ============================== =========================== ==============================
}

// -- DEVICE OPTOMIZE 2 -- //
__global__ void convolutionOnDeviceOpt2(int width, int padding, int rows, int height, int filterWidth, uint8_t *grayscalePixels, float *filter1, float *filter2, uint8_t *convoPixels)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (padding != 0 && r < 1)
        return;
    if (r > rows)
        return;

    if (r < height && c < width)
    {   
        // printf("%d ", r);
        float new_pixel_1 = 0;
        float new_pixel_2 = 0;
        for (int j = 0; j < filterWidth; j++)
        {
            for (int k = 0; k < filterWidth; k++)
            {
                int current_row = r + j - (filterWidth / 2);
                int current_column = c + k - (filterWidth / 2);
                if (current_row < 0)
                {
                    current_row = 0;
                }
                else if (current_row > height - 1)
                {
                    current_row = height - 1;
                }
                if (current_column < 0)
                {
                    current_column = 0;
                }
                else if (current_column > width - 1)
                {
                    current_column = width - 1;
                }
                int i = current_row * width + current_column;
                new_pixel_1 += grayscalePixels[i] * filter1[j * filterWidth + k];
                new_pixel_2 += grayscalePixels[i] * filter2[j * filterWidth + k];
            }
        }
        float new_pixel = sqrt(pow(new_pixel_1, 2) + pow(new_pixel_2, 2)); // sqrt(sobel_x ^ 2 + sobel_y ^ 2)
        if (new_pixel > 255)
        {
            new_pixel = 255;
        }
        else if (new_pixel < 0)
        {
            new_pixel = 0;
        }
        int i = r * width + c;
        convoPixels[i] = new_pixel;
    }
}

void seamCarvingOnDeviceOpt2(uint8_t *inPixels, char *fileName, int numChannels, int width, int height, int nStreams, float *filter1, float *filter2, int filterWidth, dim3 blockSize, uint8_t *grayscalePixels, uint8_t *convoPixels, uint8_t *minEnergies, uint8_t *backtrack, int *vMinEnergies, int *min_seam_track, uint8_t *outPixels, int* minEnergiesInt)
{
    float * d_filter1, *d_filter2;
    uint8_t * d_in, *d_grayscale, *d_convo, *d_backtrack, *d_minE, *d_out;
    int *d_vMinE;
    int * d_min_seam_track;
    // Gridsize
    dim3 gridSize(
        (width - 1) / blockSize.x + 1,
        (height - 1) / blockSize.y + 1);

    // , *d_backtrack, *d_minE, *d_vMinE, *d_minS, *d_out;
    size_t byteColor = width*height*numChannels*sizeof(uint8_t);
    size_t byte = width * height *sizeof(uint8_t);
    size_t byteOut = (width-1) * height * numChannels * sizeof(uint8_t);
    size_t byteSeam = height *sizeof(int);
    size_t filterSize = filterWidth * filterWidth * sizeof(float);
    CHECK(hipMalloc(&d_filter1, filterSize));
    CHECK(hipMalloc(&d_filter2, filterSize));
    CHECK(hipMemcpy(d_filter1, filter1, filterSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_filter2, filter2, filterSize, hipMemcpyHostToDevice));

    CHECK(hipMalloc(&d_in, byteColor));
    CHECK(hipMalloc(&d_grayscale,byte));
    CHECK(hipMalloc(&d_convo,byte));
    CHECK(hipMalloc(&d_minE,byte));
    CHECK(hipMalloc(&d_backtrack,byte));
    CHECK(hipMalloc(&d_min_seam_track,byteSeam));
    CHECK(hipMalloc(&d_out,byteOut));
    CHECK(hipMalloc(&d_vMinE,width*sizeof(int)));
    CHECK(hipMemcpy(d_vMinE, vMinEnergies, width*sizeof(int), hipMemcpyHostToDevice));

    int most_rows, last_rows;
    most_rows = (height-1)/nStreams+1;
    last_rows = height - (nStreams-1)*most_rows;
    hipStream_t *streams = (hipStream_t *) malloc(nStreams * sizeof(hipStream_t));
    for (int i = 0;i<nStreams; i++) {
        CHECK(hipStreamCreate(&streams[i]));
    }
    int done_row = 0;
    for (int i = nStreams - 1; i>=0; i--) {
        int s = nStreams - 1 - i;
        int rows = (i==0?last_rows:most_rows);
        int padding_row = height - done_row - rows;
        int paddingColor = padding_row * width * numChannels;
        size_t bytesColor = rows * width * numChannels * sizeof(uint8_t);
        CHECK(hipMemcpyAsync(d_in+paddingColor, inPixels+paddingColor, bytesColor, hipMemcpyHostToDevice, streams[s]));
        dim3 gridGray((width-1)/blockSize.x+1, (rows-1)/blockSize.y+1);
        int padding = padding_row * width;
        size_t bytes = rows * width * sizeof(uint8_t);
        grayscaleOnDevice<<<gridGray, blockSize, 0, streams[s]>>>(width, rows, d_in+paddingColor, d_grayscale+padding);
        CHECK(hipMemcpyAsync(grayscalePixels+padding, d_grayscale+padding, bytes, hipMemcpyDeviceToHost, streams[s]));

        int rows_to_conv = rows;
        int h = rows + 2;
        bytes = (rows+1) *width *sizeof(uint8_t);
        if (i==nStreams-1) {
            rows_to_conv--;
            h=rows;
            bytes = rows*width*sizeof(uint8_t);
        }
        if (i==0) {
            rows_to_conv++;
        }
        dim3 gridConv((width-1)/blockSize.x+1, (rows_to_conv-1)/blockSize.y+1);
        convolutionOnDeviceOpt2<<<gridConv, blockSize, 1, streams[s]>>>(width, padding_row, rows_to_conv, h, filterWidth, d_grayscale+padding, d_filter1, d_filter2, d_convo+padding);
        CHECK(hipMemcpyAsync(convoPixels+padding, d_convo+padding, bytes, hipMemcpyDeviceToHost, streams[s]));
        done_row+=rows;
    }

    // CHECK(cudaMemcpy(vMinEnergies, d_vMinE, width*sizeof(int), cudaMemcpyDeviceToHost));
    
    // ============================== Energy to Min Energy ==============================

    // Loop from bottom to top row
    // -> Calculate the min energy of each pixel:
    //    +) The bottom row: Just copy the energy -> min energy
    //    +) The remained above rows: Calculate min energy on the current row by using:
    //                                -) The current energy
    //                                -) The min of 3-min-energy-on-the-row-below
    // Simultaneously backtrack the direction of each pixel (which of the 3-energies-below)

    // // The bottom row
    for (int c = 0; c < width; c++)
    {
        int i = (height - 1) * width + c;
        int tempMinE = convoPixels[i];
        minEnergiesInt[i] = tempMinE * 1.0;
        backtrack[i] = 1;
    }

    // // The remained above rows
    for (int r = 1; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = (height - 1 - r) * width + c;
            int e[3] = {99999, 99999, 99999};
            if (c == 0)
            {
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            else if (c == width - 1)
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
            }
            else
            {
                e[0] = minEnergiesInt[((height - 1 - r) + 1) * width + (c - 1)];
                e[1] = minEnergiesInt[((height - 1 - r) + 1) * width + c];
                e[2] = minEnergiesInt[((height - 1 - r) + 1) * width + (c + 1)];
            }
            // uint8_t min_idx = min_e_idx(e[0], e[1], e[2]);
            uint8_t min_idx = 255;
            if (e[1] <= e[0] && e[1] <= e[2])
            {
                min_idx = 1; // return 1;
            }
            else if (e[2] <= e[0] && e[2] <= e[1])
            {
                min_idx = 2; // return 2;
            }
            else if (e[0] <= e[1] && e[0] <= e[2])
            {
                min_idx = 0; // return 0;
            }
            int tempMinE = convoPixels[i] + e[min_idx]; // Calculate 1 minimal energy base on 3-energies-below
            minEnergiesInt[i] = tempMinE * 1.0;
            backtrack[i] = min_idx; // Save the direction (which of the 3-energies-below)
        }
    }

    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int tempMinE = 0;
            tempMinE = minEnergiesInt[i];
            minEnergies[i] = tempMinE * 0.02;
        }
    }

    // // Output
    // writePnm(minEnergies, 1, width, height, concatStr("out_minEnergy", ".pnm"));
    // ============================== Extra: Vertical Min Energy (Parallel) ==============================

    // Loop through columns and calculate the sum of energies of each column
    // This will be used to help to choose which pixel to start when find the min seam later

    // Copy data
    // CHECK(cudaMemcpy(d_convoPixels, convoPixels, nBytes_convoPixels, cudaMemcpyHostToDevice)); // No need
    // Call kernel
    vMinEnergiesOnDevice<<<gridSize, blockSize>>>(width, height, d_convo, d_vMinE);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(vMinEnergies, d_vMinE, width * sizeof(int), hipMemcpyDeviceToHost));
    // ============================== Find the Min Seam ==============================
    for (int c = 0; c < width; c++)
    {
        uint8_t temp_min_e;
        int temp_min_v_e;
        if (c == 0)
        {
            temp_min_e = minEnergies[0];
            temp_min_v_e = vMinEnergies[0];
            min_seam_track[0] = 0;
        }
        else
        {
            bool is_satisfied = false;
            // ----- Just a trick -----
            if (width % 2 == 0)
            {
                if (minEnergies[c] < temp_min_e && vMinEnergies[c] <= temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            else
            {
                if (minEnergies[c] <= temp_min_e && vMinEnergies[c] < temp_min_v_e)
                {
                    is_satisfied = true;
                }
            }
            if (is_satisfied == true)
            {
                temp_min_e = minEnergies[c];
                temp_min_v_e = vMinEnergies[c];
                min_seam_track[0] = c;
            }
        }
    }

    // ------------------------------ Second step (No parallel) ------------------------------

    for (int r = 1; r < height; r++)
    {
        int prev_c = min_seam_track[r - 1];
        uint8_t direction = backtrack[(r - 1) * width + prev_c];
        if (direction == 0)
        {
            if (prev_c == 0)
            {
                min_seam_track[r] = 0;
            }
            else
            {
                min_seam_track[r] = prev_c - 1;
            }
        }
        else if (direction == 1)
        {
            min_seam_track[r] = prev_c;
        }
        else if (direction == 2)
        {
            if (prev_c == width - 1)
            {
                min_seam_track[r] = width - 1;
            }
            else
            {
                min_seam_track[r] = prev_c + 1;
            }
        }
    }
    CHECK(hipMemcpy(d_min_seam_track, min_seam_track, byteSeam, hipMemcpyHostToDevice));
    // Call kernel
    removeMinSeamOnDevice<<<gridSize, blockSize>>>(width, height, d_in, d_min_seam_track, d_out);
    CHECK(hipDeviceSynchronize());
    // Copy data
    CHECK(hipMemcpy(outPixels, d_out, byteOut, hipMemcpyDeviceToHost));

    writePnm(outPixels, 3, width - 1, height, fileName);

    for (int i = 0; i < nStreams; i++){
      CHECK(hipStreamDestroy(streams[i]));
    }
    free(streams);
    CHECK(hipFree(d_filter1));
    CHECK(hipFree(d_filter2));

    CHECK(hipFree(d_in));
    CHECK(hipFree(d_grayscale));
    CHECK(hipFree(d_convo));
    CHECK(hipFree(d_minE));
    CHECK(hipFree(d_backtrack));
    CHECK(hipFree(d_min_seam_track));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_vMinE));

}

// ======================================== MAIN ========================================
int main(int argc, char **argv)
{
    // Args
    dim3 blockSize(32, 32);
    char *fileName = argv[1];

    int mode = atoi(argv[2]);
    int numSeamsToCarve = 1;

    if (mode != 0)
    {
        numSeamsToCarve = atoi(argv[3]);
    }
    if (mode != 0 && mode != 1)
    {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    }

    // Filters
    float noFilter[] = {
        0, 0, 0,
        0, 0, 0,
        0, 0, 0};
    float identityFilter[] = {
        0, 0, 0,
        0, 1, 0,
        0, 0, 0};
    float gaussianBlur[] = {
        1.0 / 16, 2.0 / 16, 1.0 / 16,
        2.0 / 16, 4.0 / 16, 2.0 / 16,
        1.0 / 16, 2.0 / 16, 1.0 / 16};
    float laplacian[] = {
        0, -1, 0,
        -1, 4, -1,
        0, -1, 0};
    float sobel_x[] = {
        -1, 0, 1,
        -2, 0, 2,
        -1, 0, 1};
    float sobel_y[] = {
        -1, -2, -1,
        0, 0, 0,
        1, 2, 1};
    float temp = noFilter[0]; // temp
    temp = identityFilter[0]; // temp
    temp = gaussianBlur[0];   // temp
    temp = laplacian[0];      // temp
    temp = sobel_x[0];        // temp
    temp = sobel_y[0];        // temp
    temp = temp;
    int filterWidth = 3;
    float *filter1 = (float *)malloc(filterWidth * filterWidth * sizeof(float));
    float *filter2 = (float *)malloc(filterWidth * filterWidth * sizeof(float));
    for (int i = 0; i < filterWidth; i++)
    {
        for (int j = 0; j < filterWidth; j++)
        {
            filter1[i * filterWidth + j] = sobel_x[i * filterWidth + j];
            filter2[i * filterWidth + j] = sobel_y[i * filterWidth + j];
        }
    }
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_filter1), filter1, filterWidth * filterWidth * sizeof(float))); // CMEM
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(dc_filter2), filter2, filterWidth * filterWidth * sizeof(float))); // CMEM

    int numChannels, width, height;

    // Host Memories
    uint8_t *inPixels;
    readPnm(fileName, numChannels, width, height, inPixels); // Just to get raw width and height
    uint8_t *grayscalePixels = (uint8_t *)malloc(width * height * 1 * sizeof(uint8_t));
    uint8_t *convoPixels = (uint8_t *)malloc(width * height * 1 * sizeof(uint8_t));
    uint8_t *minEnergies = (uint8_t *)malloc(width * height * 1 * sizeof(uint8_t));
    int *minEnergiesInt = (int *)malloc(width * height * 1 * sizeof(int));
    uint8_t *backtrack = (uint8_t *)malloc(width * height * 1 * sizeof(uint8_t));
    int *vMinEnergies = (int *)malloc(width * sizeof(int));
    int *min_seam_track = (int *)malloc(height * sizeof(int));
    uint8_t *outPixels = (uint8_t *)malloc((width - 1) * height * numChannels * sizeof(uint8_t)); // width-1: means that the output image will be cut 1 pixel (the min seam)

    // Device Memories
    uint8_t *d_inPixels;
    uint8_t *d_grayscalePixels;
    uint8_t *d_convoPixels;
    float *d_filter1, *d_filter2;
    uint8_t *d_minEnergies, *d_backtrack;
    int *d_vMinEnergies;
    int *d_min_seam_track;
    uint8_t *d_outPixels;

    size_t nBytes_inPixels = width * height * numChannels * sizeof(uint8_t);
    size_t nBytes_grayscalePixels = width * height * 1 * sizeof(uint8_t);
    size_t nBytes_convoPixels = width * height * 1 * sizeof(uint8_t);
    size_t nBytes_filter = filterWidth * filterWidth * sizeof(float);
    size_t nBytes_minEnergies = width * height * 1 * sizeof(uint8_t);
    size_t nBytes_backtrack = width * height * 1 * sizeof(uint8_t);
    size_t nBytes_vMinEnergies = width * sizeof(int);
    size_t nBytes_min_seam_track = height * sizeof(int);
    size_t nBytes_outPixels = (width - 1) * height * numChannels * sizeof(uint8_t);

    CHECK(hipMalloc(&d_inPixels, nBytes_inPixels));
    CHECK(hipMalloc(&d_grayscalePixels, nBytes_grayscalePixels));
    CHECK(hipMalloc(&d_convoPixels, nBytes_convoPixels));
    CHECK(hipMalloc(&d_filter1, nBytes_filter));
    CHECK(hipMalloc(&d_filter2, nBytes_filter));
    CHECK(hipMalloc(&d_minEnergies, nBytes_minEnergies));
    CHECK(hipMalloc(&d_backtrack, nBytes_backtrack));
    CHECK(hipMalloc(&d_vMinEnergies, nBytes_vMinEnergies));
    CHECK(hipMalloc(&d_min_seam_track, nBytes_min_seam_track));
    CHECK(hipMalloc(&d_outPixels, nBytes_outPixels));

    if (mode != 0)
    {
        // =========================================================================
        GpuTimer timer;
        timer.Start();
        // --------------------- SEAM CARVING - Remove N seams ---------------------
        printf("Seam Carving is running...\n");

        int last_i = 0;
        for (int i = 0; i < numSeamsToCarve; i++)
        {
            // Read input RGB image file
            readPnm(fileName, numChannels, width, height, inPixels);
            nBytes_inPixels = width * height * numChannels * sizeof(uint8_t);
            // SEAM CARVING - Remove 1 seam
            if (mode == 1) // Host
            {
                seamCarvingOnHost(inPixels, fileName, numChannels, width, height, filter1, filter2, filterWidth, blockSize, grayscalePixels, convoPixels, minEnergies, backtrack, vMinEnergies, min_seam_track, outPixels, minEnergiesInt);
            }
            else if (mode == 2) // Device
            {
                seamCarvingOnDevice(inPixels, fileName, numChannels, width, height, filter1, filter2, filterWidth, blockSize, grayscalePixels, convoPixels, minEnergies, backtrack, vMinEnergies, min_seam_track, outPixels, d_inPixels, nBytes_inPixels, d_grayscalePixels, nBytes_grayscalePixels, d_convoPixels, nBytes_convoPixels, d_filter1, d_filter2, nBytes_filter, d_backtrack, nBytes_backtrack, d_minEnergies, nBytes_minEnergies, d_vMinEnergies, nBytes_vMinEnergies, d_min_seam_track, nBytes_min_seam_track, d_outPixels, nBytes_outPixels, minEnergiesInt);
            }
            else if (mode == 3) // Device (Optimize 1)
            {
                seamCarvingOnDeviceOpt1(inPixels, fileName, numChannels, width, height, filterWidth, blockSize, grayscalePixels, convoPixels, minEnergies, backtrack, vMinEnergies, min_seam_track, outPixels, d_inPixels, nBytes_inPixels, d_grayscalePixels, nBytes_grayscalePixels, d_convoPixels, nBytes_convoPixels, d_backtrack, nBytes_backtrack, d_minEnergies, nBytes_minEnergies, d_vMinEnergies, nBytes_vMinEnergies, d_min_seam_track, nBytes_min_seam_track, d_outPixels, nBytes_outPixels, minEnergiesInt);
            }
            else if (mode == 4) // Device (Optimize 2)
            {
                // printf("> Not implemented yet 🙀");
                seamCarvingOnDeviceOpt2(inPixels, fileName, numChannels, width, height, atoi(argv[6]), filter1, filter2, filterWidth, blockSize, grayscalePixels, convoPixels, minEnergies, backtrack, vMinEnergies, min_seam_track, outPixels, minEnergiesInt);
            }
            // printf("> i = %d (%d x %d) finished\n", i, width, height);
            last_i = i;
        }
        printf("> Removed %d seams\n", last_i + 1);

        // -------------------------------------------------------------------------
        timer.Stop();
        float time = timer.Elapsed();
        if (mode == 1) // Host
        {
            printf("> Using HOST\n");
        }
        else if (mode == 2) // Device
        {
            printf("> Using DEVICE\n");
        }
        else if (mode == 3) // Device (Optimize 1)
        {
            printf("> Using DEVICE (OPTIMIZE 1)\n");
        }
        else if (mode == 4) // Device (Optimize 2)
        {
            printf("> Using DEVICE (OPTIMIZE 2)\n");
        }
        printf("> Processing time: %f ms\n", time);
        // =========================================================================
        printf(">>> Completed!\n");
    }
    else
    {
        readPnm(fileName, numChannels, width, height, inPixels);
        nBytes_inPixels = width * height * numChannels * sizeof(uint8_t);
        seamCarvingOnHost(inPixels, fileName, numChannels, width, height, filter1, filter2, filterWidth, blockSize, grayscalePixels, convoPixels, minEnergies, backtrack, vMinEnergies, min_seam_track, outPixels, minEnergiesInt);
    }

    // ----- Free Host Memories
    free(inPixels);
    free(filter1);
    free(filter2);
    free(grayscalePixels);
    free(convoPixels);
    free(minEnergies);
    free(minEnergiesInt);
    free(backtrack);
    free(vMinEnergies);
    free(min_seam_track);
    free(outPixels);
    // ----- Free Device Memories
    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayscalePixels));
    CHECK(hipFree(d_convoPixels));
    CHECK(hipFree(d_filter1));
    CHECK(hipFree(d_filter2));
    CHECK(hipFree(d_minEnergies));
    CHECK(hipFree(d_backtrack));
    CHECK(hipFree(d_vMinEnergies));
    CHECK(hipFree(d_min_seam_track));
    CHECK(hipFree(d_outPixels));
}
